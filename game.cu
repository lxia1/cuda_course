#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define ROWS 6
#define COLS 7

// Utility function to check if a move leads to a win
__device__ bool check_win(int board[ROWS][COLS], int player) {
    for (int row = 0; row < ROWS; ++row) {
        for (int col = 0; col < COLS; ++col) {
            if (col + 3 < COLS && 
                board[row][col] == player && 
                board[row][col + 1] == player && 
                board[row][col + 2] == player && 
                board[row][col + 3] == player) {
                return true;
            }
            if (row + 3 < ROWS &&
                board[row][col] == player &&
                board[row + 1][col] == player &&
                board[row + 2][col] == player &&
                board[row + 3][col] == player) {
                return true;
            }
            if (row + 3 < ROWS && col + 3 < COLS &&
                board[row][col] == player &&
                board[row + 1][col + 1] == player &&
                board[row + 2][col + 2] == player &&
                board[row + 3][col + 3] == player) {
                return true;
            }
            if (row + 3 < ROWS && col - 3 >= 0 &&
                board[row][col] == player &&
                board[row + 1][col - 1] == player &&
                board[row + 2][col - 2] == player &&
                board[row + 3][col - 3] == player) {
                return true;
            }
        }
    }
    return false;
}

// Kernel to make a move for a given player (randomly select a column)
__global__ void make_move(int* board, int player, int* move_made, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    bool move_found = false;

    while (!move_found) {
        int col = hiprand(&states[idx]) % COLS;
        // Check if there's an empty spot in the selected column
        for (int row = ROWS - 1; row >= 0; --row) {
            if (board[row * COLS + col] == 0) {  // Empty spot found
                board[row * COLS + col] = player;
                *move_made = col;
                move_found = true;  // Valid move made
                break;
            }
        }
    }
}

// Initialize the random states
__global__ void init_random(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

// Host function to print the board with Player 1 as 'X' and Player 2 as 'O'
void print_board(int board[ROWS][COLS]) {
    for (int row = 0; row < ROWS; ++row) {
        for (int col = 0; col < COLS; ++col) {
            if (board[row][col] == 1) {
                std::cout << "X ";  // Player 1 move
            } else if (board[row][col] == 2) {
                std::cout << "O ";  // Player 2 move
            } else {
                std::cout << ". ";  // Empty space
            }
        }
        std::cout << std::endl;
    }
}

// Host function to check for win condition on the CPU
bool check_win_host(int board[ROWS][COLS], int player) {
    for (int row = 0; row < ROWS; ++row) {
        for (int col = 0; col < COLS; ++col) {
            if (col + 3 < COLS && 
                board[row][col] == player && 
                board[row][col + 1] == player && 
                board[row][col + 2] == player && 
                board[row][col + 3] == player) {
                return true;
            }
            if (row + 3 < ROWS &&
                board[row][col] == player &&
                board[row + 1][col] == player &&
                board[row + 2][col] == player &&
                board[row + 3][col] == player) {
                return true;
            }
            if (row + 3 < ROWS && col + 3 < COLS &&
                board[row][col] == player &&
                board[row + 1][col + 1] == player &&
                board[row + 2][col + 2] == player &&
                board[row + 3][col + 3] == player) {
                return true;
            }
            if (row + 3 < ROWS && col - 3 >= 0 &&
                board[row][col] == player &&
                board[row + 1][col - 1] == player &&
                board[row + 2][col - 2] == player &&
                board[row + 3][col - 3] == player) {
                return true;
            }
        }
    }
    return false;
}

// Function to wait for key press
void wait_for_keypress() {
    std::cout << "Press Enter to continue..." << std::endl;
    std::cin.get();  // Wait for the user to press Enter
}

// Host function to play the game using two GPUs
void play_game() {
    int board[ROWS][COLS] = {0};  // Initialize empty board
    int* d_board[2];
    int move_made[2], *d_move_made[2];
    hiprandState* d_states[2];
    int current_player = 1;

    // Allocate memory for both GPUs
    for (int device = 0; device < 2; ++device) {
        hipSetDevice(device);
        hipMalloc(&d_board[device], ROWS * COLS * sizeof(int));
        hipMalloc(&d_move_made[device], sizeof(int));
        hipMalloc(&d_states[device], sizeof(hiprandState) * 1);
        hipMemcpy(d_board[device], board, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);
        init_random<<<1, 1>>>(d_states[device], time(0) + device);  // Different seeds for each GPU
    }

    int turn = 1;
    while (true) {
        int device = current_player - 1;  // 0 for GPU 1, 1 for GPU 2

        // Set the correct device for the current player
        hipSetDevice(device);

        // GPU makes a move
        make_move<<<1, 1>>>(d_board[device], current_player, d_move_made[device], d_states[device]);

        // Copy move result and board back to host
        hipMemcpy(&move_made[device], d_move_made[device], sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(board, d_board[device], ROWS * COLS * sizeof(int), hipMemcpyDeviceToHost);

        // Synchronize board state to the other GPU
        int other_device = (current_player == 1) ? 1 : 0;
        hipSetDevice(other_device);
        hipMemcpy(d_board[other_device], board, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);

        // Print the current board state
        std::cout << "Trun "<< turn <<": Player " << current_player << " (GPU " << device << ") made a move in column " << move_made[device] << std::endl;
        print_board(board);


        // Check for a win
        if (check_win_host(board, current_player)) {
            std::cout << "Player " << current_player << " (GPU " << device << ") wins!" << std::endl;
            break;
        }

        // Alternate players
        current_player = (current_player == 1) ? 2 : 1;

        // Wait for key press after the turn
        wait_for_keypress();

        turn++;
    }

    // Clean up
    for (int device = 0; device < 2; ++device) {
        hipSetDevice(device);
        hipFree(d_board[device]);
        hipFree(d_move_made[device]);
        hipFree(d_states[device]);
    }
}

int main() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    printf("Number of GPU Devices: %d\n", nDevices);

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device Compute Major: %d Minor: %d\n", prop.major, prop.minor);
        printf("  Max Thread Dimensions: [%d][%d][%d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Number of Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Device Clock Rate (KHz): %d\n", prop.clockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Registers Per Block: %d\n", prop.regsPerBlock);
        printf("  Registers Per Multiprocessor: %d\n", prop.regsPerMultiprocessor);
        printf("  Shared Memory Per Block: %zu\n", prop.sharedMemPerBlock);
        printf("  Shared Memory Per Multiprocessor: %zu\n", prop.sharedMemPerMultiprocessor);
        printf("  Total Constant Memory (bytes): %zu\n", prop.totalConstMem);
        printf("  Total Global Memory (bytes): %zu\n", prop.totalGlobalMem);
        printf("  Warp Size: %d\n", prop.warpSize);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        // You can set the current chosen device property based on tracked min/max values
        
    }

    if (nDevices>=2) {
        play_game();
        return 0;
    }
    else {
        std::cout<< "You need 2 GPU devices to run the game."<<std::endl;
        return -1;
    }
}
